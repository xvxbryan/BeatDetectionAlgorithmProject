#include "hip/hip_runtime.h"
//gpubpm.cu
/*
 Team: Seahawks

 Program Description: Determines how many beats per minute are in a song using
 a list of frequencies that must be provided by a file in
 the command-line.

 The algorithm to determine the beats per minute were provided
 by Marco Ziccardi. His beat detection algorithm can be found in
 the link below:

 http://mziccard.me/2015/05/28/beats-detection-algorithms-1/
 */

#include <emmintrin.h>
#include <sys/time.h>
#include <stdio.h>
#include<stdlib.h>
#include<limits.h>

#define SAMPS_IN_SONG 		9281536
#define SAMPLE_RATE 		44100
#define SAMPLES_PER_MIN 	SAMPLE_RATE * 60
#define UNCALCULATED_SAMPS 	68
#define SAMPLES_PER_BLOCK 	1024
#define BLOCKS_PER_SECOND 	SAMPLE_RATE / SAMPLES_PER_BLOCK
#define C_MULTIPLIER 		-0.0000015
#define C_ADDER 			1.5142857
#define BLOCKS 43

/*Prototypes*/
int initialize(float *, int, char**);
__device__ void gpuSquared(float *, int);
__device__ void gpuCalcInstantEnergies(float *, float *);
__global__ void getInstantEnergies(float *, float *, int);
void calcBPM(float*, int);
float getAvgEnergy(float *, int);
double getVariance(float, float *, int);
double getSoil(float, float);
int calcBeats(float *, float, int);
int getBeats(float * ejs, int totalFrequencies);



int initialize(float *frequency, int argc, char** argv) {
	/*Variables*/
	FILE * file;
	file = fopen(argv[argc - 1], "r");

	int totalFrequencies = 0;
	while (fscanf(file, "%f", &frequency[totalFrequencies]) != EOF) {
		totalFrequencies++;
	}
	return totalFrequencies;
}


/*
	The GPU kernal gpuSquared(float* frequency, int totalFrequencies) accepts
	two parameters for the input vector. Each thread squares the given array's
	elements and multiplies it by two in its respective place. The size of the
	array is passed into the kernal as numOfFrequencies. The result is saved
	into the array's original position.

	The original equation provided by Marco Ziccardi takes the left and right
	frequencies at the same index, squares each of the elements and adds them
	together. However that is when the song is in stereo. This kernal only
	supports songs in mono.

	@parameters array is an array of float elements that are to be squared.
	@parameters numOfFrequencies is the number of elements in the array.
*/
__device__ void gpuSquared(float frequency[], int totalFrequencies) {
	/*Variables*/
	int element = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (element < totalFrequencies)
		frequency[element] = 2 * (frequency[element] * frequency[element]);
}


/*
	Function getBlocks(float ej[], float sampleArray[], long samplesPerBlock, 
	long samples) computes the energy of a block. A block is made up of 1024
	samples in mono. The energy in a block is computed by summing a block
	of sampleArray and returning the result into the given array ej.
	The equation provided by Marco Ziccardi is:

		  1024
	Ej =   ∑  sampleArray[i]
		 i = 0

	@parameters ej is an array of floats that returns the result of the 
	summation.
	@parameters sampleArray is an array of floats that contains the results
		of each left and right frequency squared and multiplied by two
		in each index.
	@parameters samplesPerBlock indicates indicates block we are at, must
		be a multiple of 1024.
	@parameters samples is the current sample per second.
	@return ej which is an array containing the energies of the blocks.
*/
__device__ void gpuCalcInstantEnergies(float frequency[], float instantEnergy[]) {
	/*Variables*/
	unsigned int tid = threadIdx.x;
	unsigned int element = blockIdx.x * blockDim.x + tid;

	/*The last 68 samples of a second don't get computed*/
	unsigned int offset = blockIdx.x / BLOCKS_PER_SECOND;
	offset *= UNCALCULATED_SAMPS;

	instantEnergy[element] = frequency[element];
	__syncthreads();

	for (unsigned int s = 1; s < SAMPLES_PER_BLOCK; s *= 2) {
		if (tid % (2 * s) == 0) {
			instantEnergy[element + offset] += instantEnergy[element + s
					+ offset];
		}
		__syncthreads();
	}

	if (tid == 0) {
		frequency[blockIdx.x] = instantEnergy[element + offset];
	}
}

__global__ void getInstantEnergies(float * frequencies, float * energy, int samples) {
	gpuSquared(frequencies, samples);
	__syncthreads();
	
	gpuCalcInstantEnergies(frequencies, energy);
	__syncthreads();
}


/*
	Function calcBPM (float *samples, int totalFrequencies) allocates
	GPU memory and transfers the data between the CPU and GPU to get the instant
	energy of each block. Once it has the instant energies of all the elements it
	transfers the data between the GPU to the CPU and calls the following functions
	get the beat count of the song. A simple formula is then applied to this beat
	count to calculate the BPM of the song.

	@parameters samples is an array of float elements that are to be squared.
	@parameters totalFrequencies is the number of elements in the array.
*/
void calcBPM(float* samples, int totalFrequencies) {
	/*Variables*/
	int numThreads = 1024;
	int numCores = totalFrequencies / 1024 + 1;
	int bpm = 0;
	int beats = 0;

	float* gpuA;
	hipMalloc(&gpuA, totalFrequencies * sizeof(float));
	hipMemcpy(gpuA, samples, totalFrequencies * sizeof(float),
			hipMemcpyHostToDevice);

	float* gpuB;
	hipMalloc(&gpuB, totalFrequencies * sizeof(float));

	getInstantEnergies<<<numCores, numThreads>>> (gpuA, gpuB, totalFrequencies);

	hipMemcpy(samples, gpuA, totalFrequencies * sizeof(float),
			hipMemcpyDeviceToHost);
	hipFree(&gpuA);
	hipFree(&gpuB);

	/*Samples contain the instant energies*/
	beats = getBeats(samples, totalFrequencies);

	bpm = (int) ((beats * SAMPLES_PER_MIN) / totalFrequencies);
	printf("BPM = %d\n", bpm);

}


/*
	Function getAvgEnergy(float ej[]) computes the average window energy with
	a sample rate of 44100 and 43 blocks per current window, which slightly
	more than 1 second of music. The equation provided by Marco Ziccardi is:

					 42
	avg(E) = (1/43)  ∑  ej[i]
					i = 0

	@parameters ej is an array of floats containing the energy computed in 
		each block.
	@return avg is the computed average energy in the current window made
		up of 43 blocks.
 */
float getAvgEnergy(float * ejs, int currentSec) {
	/*Variables*/
	int currentEnergy = BLOCKS_PER_SECOND * currentSec;
	int lastEnergy = currentEnergy + BLOCKS_PER_SECOND;
	float avg = 0;

	while (currentEnergy < lastEnergy) {
		avg += ejs[currentEnergy];
		currentEnergy++;
	}
	avg = avg / BLOCKS;

	return avg;
}


/*
	Function getVariance(float ej[], float avg) computes the variance inside
	a window of blocks. The bigger the variance, the more likely a block will
	be considered a beat. The equation provided by Marco Ziccardi is:

					42
	var(E) = (1/43) ∑	(avg(E) - Ej)^2
				  i = 0

	@parameters ej is an array of floats containing the energy computed in
		each block.
	@parameters avg is the average energy in the current window made up of
		43 blocks.
	@return variance is the calculated variance of a window of blocks.
*/
double getVariance(float avg, float * ejs, int currentSec) {
	/*Variables*/
	float var = 0.0;
	int currentEnergy = BLOCKS_PER_SECOND * currentSec;
	int lastEnergy = currentEnergy + BLOCKS_PER_SECOND;
	float temp;

	while (currentEnergy < lastEnergy) {
		temp = avg - ejs[currentEnergy];
		var += pow(temp, 2.0);
		currentEnergy++;
	}
	var /= BLOCKS;

	return var;
}


/*
	Function getSoil(float var, float avg) computes the linear regression of the
	energy variance that lowers the impact of the variance provided by 
	Marco Ziccardi that is used to determine if a beat has occurred.
 
	C = −0.0000015 * var(E) + 1.5142857
 
	@parameters var is a float that contains the current variance.
	@parameters avg is a float that contains the average energy of the current Ej.
	@return soil returns the computed linear regression of the current energy
		variance.
 */
double getSoil(float var, float avg) {
	/*Variables*/
	float soil = 0.0;

	soil = (var * C_MULTIPLIER) + C_ADDER;
	soil *= avg;

	return soil;
}

/*
	Function calcBeats(float * ejs, float soil, int currentSec) detects
	a peak if the instant energy is bigger than c * avg(E).
	If a peak is detected 4 times in a row it is counted as a beat.

	@parameters ejs is a pointer to an array of floats that contains
		the computed ejs.
	@parameters soil contains the linear regression of the current
		energy variance.
*/
int calcBeats(float * ejs, float soil, int currentSec) {
	/*Variables*/
	int beats = 0;
	int peakCounter = 0;
	float energy = 0.0;
	int currentEnergy = currentSec * BLOCKS_PER_SECOND;
	float lastEnergy = currentEnergy + BLOCKS_PER_SECOND;

	while (currentEnergy < lastEnergy) {
		energy = ejs[currentEnergy];
		
		if (energy > soil) {
			peakCounter++;
			if (peakCounter == 4) {
				beats++;
				peakCounter = 0;
			}
		} else {
			peakCounter = 0;
		}
		currentEnergy++;
	}

	return beats;
}


/*
	Function getBeats(float * ejs, int totalFrequencies) computes per
	second the neccessary parameters to determine whether
	a beat has occured, and increments the beat count if beats are found.
 
	@parameters ejs is the array holding the instant energies
	@parameters totalFrequencies is the number of samples in the song
 */
int getBeats(float * ejs, int totalFrequencies) {
	/*Variables*/
	int beats = 0;
	int secsInSong = totalFrequencies / SAMPLE_RATE;
	float avg = 0.0;
	float var = 0.0;
	float soil = 0.0;

	int currentSec = 0;

	while (currentSec <= secsInSong) {
		avg = getAvgEnergy(ejs, currentSec);
		var = getVariance(avg, ejs, currentSec);
		soil = getSoil(var, avg);

		beats += calcBeats(ejs, soil, currentSec);
		currentSec++;
	}

	return beats;
}

int main(int argc, char** argv) {
	/*Start clock*/
	int msec = 0;
	clock_t start = clock(), diff;

	/*Variables*/
	int frequenciesInSong = 0;
	float* frequencies = (float*) malloc(INT_MAX * sizeof(float));
	printf("Starting\n");
	
	frequenciesInSong = initialize(frequencies, argc, argv);

	calcBPM(frequencies, frequenciesInSong);

	free(frequencies);

	/*Calculate Time*/
	diff = clock() - start;
	msec = diff * 1000 / CLOCKS_PER_SEC;
	printf("Time taken %d seconds %d milliseconds\n", msec / 1000, msec % 1000);

}
